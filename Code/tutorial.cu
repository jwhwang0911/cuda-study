#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>

#define DIV_EPSILON 1e-5f

__global__ void WeightedAverageForward(
    const int width,           //
    const int height,          //
    const int kernelWidth,     //
    const int halfKernelWidth, //
    const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> input,
    const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> weights,
    torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> output)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
        return;

    // Zero initializations
    output[0][y][x] = 0;
    output[1][y][x] = 0;
    output[2][y][x] = 0;

    // Iterate the kernel (v: y-axis, u: x-axis)
    float sumW = 0;
    for (int v = -halfKernelWidth; v <= halfKernelWidth; ++v)
    {
        int vy = v + y;
        if (vy < 0 || vy >= height)
            continue;
        for (int u = -halfKernelWidth; u <= halfKernelWidth; ++u)
        {
            int ux = u + x;
            if (ux < 0 || ux >= width)
                continue;

            // weight index
            int ind = (v + halfKernelWidth) * kernelWidth + (u + halfKernelWidth);

            output[0][y][x] += input[0][vy][ux] * weights[ind][y][x];
            output[1][y][x] += input[1][vy][ux] * weights[ind][y][x];
            output[2][y][x] += input[2][vy][ux] * weights[ind][y][x];
            sumW += weights[ind][y][x];
        }
    }
    
    float invSumW = 1 / fmaxf(sumW, DIV_EPSILON);
    output[0][y][x] *= invSumW;
    output[1][y][x] *= invSumW;
    output[2][y][x] *= invSumW;
}

// A wrapper function that launches the kernel.
torch::Tensor launchWeightedAverageForward(
    torch::Tensor input,  //
    torch::Tensor weights //
)
{
    const int width = input.size(2);
    const int height = input.size(1);
    dim3 dimBlock(16, 16);
    dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);

    torch::Tensor output = torch::zeros_like(input);
    const uint32_t kernelWidth = (uint32_t)sqrt((float)weights.size(0));
    const uint32_t halfKernelWidth = (uint32_t)(kernelWidth / 2);

    WeightedAverageForward<<<dimGrid, dimBlock>>>(
        width, height,   // image size
        kernelWidth,     // kernel width
        halfKernelWidth, // half kernel width
        input.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
        weights.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
        output.packed_accessor32<float, 3, torch::RestrictPtrTraits>());

    hipDeviceSynchronize();

    return output;
}

__global__ void WeightedAverageBackward(
    const int width,           //
    const int height,          //
    const int kernelWidth,     //
    const int halfKernelWidth, //
    const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> input,
    const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> weights,
    const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> gradPrev,
    torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> gradWeights)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
        return;

    // Iterate the kernel (v: y-axis, u: x-axis)
    static const float factor = 1.f / 3.f;
    for (int v = -halfKernelWidth; v <= halfKernelWidth; ++v)
    {
        int vy = v + y;
        if (vy < 0 || vy >= height)
            continue;
        for (int u = -halfKernelWidth; u <= halfKernelWidth; ++u)
        {
            int ux = u + x;
            if (ux < 0 || ux >= width)
                continue;

            // weight index
            int ind = (v + halfKernelWidth) * kernelWidth + (u + halfKernelWidth);

            gradWeights[ind][y][x] = input[0][vy][ux] * gradPrev[0][y][x]   //
                                     + input[1][vy][ux] * gradPrev[1][y][x] //
                                     + input[2][vy][ux] * gradPrev[2][y][x];
        }
    }
}

// A wrapper function that launches the kernel.
std::vector<torch::Tensor> launchWeightedAverageBackward(
    torch::Tensor input,   // [3, H, W]
    torch::Tensor weights, // [KxK, H, W]
    torch::Tensor gradPrev // [3, H, W]
)
{
    const int width = input.size(2);
    const int height = input.size(1);
    dim3 dimBlock(16, 16);
    dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);

    // [KxK, H, W]
    torch::Tensor gradWeights = torch::zeros(
        {weights.size(0) /*kernel*/, height, width}, // Shape
        torch::TensorOptions().device(torch::kCUDA)  // Device
    );
    const uint32_t kernelWidth = (uint32_t)sqrt((float)weights.size(0));
    const uint32_t halfKernelWidth = (uint32_t)(kernelWidth / 2);

    WeightedAverageBackward<<<dimGrid, dimBlock>>>(
        width, height,   // image size
        kernelWidth,     // kernel width
        halfKernelWidth, // half kernel width
        input.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
        weights.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
        gradPrev.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
        gradWeights.packed_accessor32<float, 3, torch::RestrictPtrTraits>());

    // Return empty grad for input
    return {torch::Tensor(), gradWeights};
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m)
{
    m.def("forward", &launchWeightedAverageForward, "Weighted average forward (CUDA)");
    m.def("backward", &launchWeightedAverageBackward, "Weighted average backward (CUDA)");
}