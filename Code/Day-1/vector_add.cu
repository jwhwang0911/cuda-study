#include <hip/hip_runtime.h>
#include <iostream>

// CUDA 벡터 덧셈 커널
__global__ void vector_add(float* A, float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 1000;
    size_t size = N * sizeof(float);

    // 호스트 메모리 할당 및 초기화
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    for (int i = 0; i < N; i++) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    // 디바이스 메모리 할당
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // 호스트에서 디바이스로 데이터 복사
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // CUDA 커널 호출
    int threads_per_block = 256;
    int blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;
    vector_add<<<blocks_per_grid, threads_per_block>>>(d_A, d_B, d_C, N);

    // 결과를 디바이스에서 호스트로 복사
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // 결과 검증
    for (int i = 0; i < N; i++) {
        if (h_C[i] != h_A[i] + h_B[i]) {
            std::cerr << "Mismatch at index " << i << "!" << std::endl;
            break;
        }
    }

    // 메모리 해제
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    std::cout << "Done!" << std::endl;
    return 0;
}
